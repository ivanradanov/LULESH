#include <hip/hip_runtime.h>

#include <cstdio>
#include <fstream>
#include <iostream>
#include <map>
#include <memory>
#include <string>
#include <string_view>
#include <vector>

#define CUDA_CHECK(err)                                                  \
  do {                                                                   \
    hipError_t err_ = (err);                                            \
    if (err_ != hipSuccess) {                                           \
      std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__); \
      throw std::runtime_error("CUDA error");                            \
    }                                                                    \
  } while (0)

struct MemRedAnalysisParser {
  static constexpr std::string_view ANALYSIS_FILE = "./.memred.memory.analysis.out";

  struct KernelInfo {
    std::string funcName;
    std::string memoryEffect;
    std::vector<std::pair<int, std::string>> ptrArgInfos;
  };

  std::map<std::string, KernelInfo> funcNameToKernelInfoMap;

  MemRedAnalysisParser() {
    std::ifstream in(ANALYSIS_FILE.data());
    std::string ignore;
    while (in) {
      KernelInfo kernelInfo;

      // Read the function name
      // Example: "Function void CalcMinDtOneBlock<1024>(double*, double*, double*, double*, int) (@_Z17CalcMinDtOneBlockILi1024EEvPdS0_S0_S0_i):"
      char ch;
      do {
        in.read(&ch, 1);
      } while (in && ch != '@');
      if (!in) break;

      std::string funcName;
      in >> funcName;
      funcName.erase(funcName.size() - 2);
      kernelInfo.funcName = funcName;

      // Read the function's memory effect
      // Example: "Memory Effect: ArgMemOnly"
      std::string memoryEffect;
      in >> ignore >> ignore >> memoryEffect;
      kernelInfo.memoryEffect = memoryEffect;

      // Read the argument information
      // Example: "Arg #0:	Effect: ReadOnly  Capture: No"
      while (1) {
        std::string argumentKeyword;
        in >> argumentKeyword;
        if (argumentKeyword == "Function" || !in)
          break;
        if (argumentKeyword != "Arg") {
          abort();
        }

        in.read(&ch, 1);  // ' '
        in.read(&ch, 1);  // '#'

        size_t argumentIndex;
        in >> argumentIndex;

        std::string ptrArgEffect;

        // :  Effect: ReadOnly Capture: No
        in >> ignore >> ignore >> ptrArgEffect >> ignore >> ignore;

        kernelInfo.ptrArgInfos.push_back({argumentIndex, ptrArgEffect});
      }

      funcNameToKernelInfoMap[funcName] = kernelInfo;
    }
  }
};

void analyzeGraph(hipGraph_t graph) {
  MemRedAnalysisParser analysisParser;

  size_t numNodes;
  CUDA_CHECK(hipGraphGetNodes(graph, nullptr, &numNodes));
  auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  CUDA_CHECK(hipGraphGetNodes(graph, nodes.get(), &numNodes));

  for (size_t i = 0; i < numNodes; i++) {
    std::cout << "Node " << i << ":" << std::endl;

    hipGraphNode_t u = nodes[i];
    hipKernelNodeParams params;
    CUDA_CHECK(hipGraphKernelNodeGetParams(u, &params));

    const char *funcName;
    CUDA_CHECK(cudaFuncGetName(&funcName, params.func));
    std::string s(funcName);
    if (analysisParser.funcNameToKernelInfoMap.count(s) == 0) {
      std::cerr << "Could not find kernel " << s << std::endl;
      abort();
    }

    auto kernelInfo = analysisParser.funcNameToKernelInfoMap[s];

    std::cout << "  Func Name: " << kernelInfo.funcName << std::endl;

    for (const auto &[index, effect] : kernelInfo.ptrArgInfos) {
      std::cout << "  Arg #" << index
                << " : Effect: " << effect
                << " Value: " << (*static_cast<int **>(params.kernelParams[index]))
                << std::endl;
    }
  }
}

__global__ void foo(int a, char b, long long c, int *x, int *y) {
  *x = *y;
  printf("[foo] x -> %p\n", x);
  printf("[foo] y -> %p\n", y);
}

int main() {
  int *x, *y;
  CUDA_CHECK(hipMallocManaged(&x, sizeof(int)));
  CUDA_CHECK(hipMallocManaged(&y, sizeof(int)));
  printf("x -> %p\n", x);
  printf("y -> %p\n", y);
  *x = 0;
  *y = 1;

  hipStream_t s;
  CUDA_CHECK(hipStreamCreate(&s));
  CUDA_CHECK(hipStreamBeginCapture(s, hipStreamCaptureModeGlobal));
  foo<<<1, 1, 0, s>>>(1, 2, 3, x, y);

  hipGraph_t g;
  CUDA_CHECK(hipStreamEndCapture(s, &g));

  analyzeGraph(g);

  return 0;
}